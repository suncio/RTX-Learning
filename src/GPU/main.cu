#include "hip/hip_runtime.h"
#include <>
#include <hip/hip_runtime.h>

#include <iostream>
#include <time.h>

#include "Math/Vector3.h"
#include "Ray.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

/*
__device__ Color ray_color(const Ray& r, const Color& background, const Hittable& world, int depth)
{
	HitRecord rec;

	// If we've exceeded the ray bounce limit, no more light is gathered.
	if (depth <= 0)
		return Color(0, 0, 0);

	// If the ray hits nothing, return the background color.
	if (!world.hit(r, 0.001, infinity, rec))
		return background;

	Ray scattered;
	Color attenuation;
	Color emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.position);

	if (!rec.mat_ptr->scatter(r, rec, attenuation, scattered))
		return emitted;

	return emitted + attenuation * ray_color(scattered, background, world, depth - 1);
}
*/

__device__ Color ray_color(const Ray& r)
{
	Vector3 unit_direction = r.getDirection().getNormalied();
	float t = 0.5f * (unit_direction.y + 1.0f);
	return (1.0f - t) * Vector3(1.0, 1.0, 1.0) + t * Vector3(0.5, 0.7, 1.0);
}

__global__ void render(Vector3 *fb, int max_x, int max_y, 
	Vector3 lower_left_corner, Vector3 horizontal, Vector3 vertical, Vector3 origin)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;
	float u = float(i) / float(max_x);
	float v = float(j) / float(max_y);
	Ray r(origin, lower_left_corner + u * horizontal + v * vertical);
	fb[pixel_index] = ray_color(r);
}

int main() {
	int nx = 1200;
	int ny = 600;
	int tx = 8;
	int ty = 8;

	std::cerr << "Rendering a " << nx << "x" << ny << " image ";
	std::cerr << "in " << tx << "x" << ty << " blocks.\n";

	int num_pixels = nx * ny;
	size_t fb_size = num_pixels * sizeof(Vector3);

	// allocate FB
	Vector3 *fb;
	checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

	clock_t start, stop;
	start = clock();
	// Render our buffer
	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);
	render<<<blocks, threads >>>(fb, nx, ny,
		Vector3(-2.0, -1.0, -1.0),
		Vector3(4.0, 0.0, 0.0),
		Vector3(0.0, 2.0, 0.0),
		Vector3(0.0, 0.0, 0.0)
		);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	stop = clock();
	double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
	std::cerr << "took " << timer_seconds << " seconds.\n";

	// Output FB as Image
	std::cout << "P3\n" << nx << " " << ny << "\n255\n";
	for (int j = ny - 1; j >= 0; j--) {
		for (int i = 0; i < nx; i++) {
			size_t pixel_index = j * nx + i;
			int ir = int(255.99 * fb[pixel_index].x);
			int ig = int(255.99 * fb[pixel_index].y);
			int ib = int(255.99 * fb[pixel_index].z);
			std::cout << ir << " " << ig << " " << ib << "\n";
		}
	}

	checkCudaErrors(hipFree(fb));
}