#include "hip/hip_runtime.h"
#include <>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <time.h>
#include <float.h>

#include "Math/Vector3.h"
#include "Ray.h"
#include "Hittable.h"
#include "Camera.h"
#include "Material.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "External/stb_image_write.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

// Manually limit the depth to be 50
__device__ Color ray_color(const Ray& r, Hittable **world, hiprandState *local_rand_state)
{
	Ray cur_ray = r;
	Vector3 cur_attenuation = Vector3(1.0f, 1.0f, 1.0f);
	int depth = 50;
	
	for (int i = 0; i < depth; i++)
	{
		HitRecord rec;
		if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec))
		{
			Ray scattered;
			Vector3 attenuation;
			if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state))
			{
				cur_attenuation = cur_attenuation * attenuation;
				cur_ray = scattered;
			}
			else
			{
				return Vector3(0.0f, 0.0f, 0.0f);
			}
		}
		else
		{
			Vector3 unit_direction = r.getDirection().getNormalied();
			float t = 0.5f * (unit_direction.y + 1.0f);
			Vector3 c = (1.0f - t) * Vector3(1.0, 1.0, 1.0) + t * Vector3(0.5, 0.7, 1.0);
			return cur_attenuation * c;
		}
	}
	
	return Vector3(0.0f, 0.0f, 0.0f); // exceed recursion
}

__global__ void rand_init(hiprandState *rand_state)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		hiprand_init(2020, 0, 0, rand_state);
	}
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = i + j * max_x;
	// Original: Each thread gets same seed, a different sequence number, no offset
	// hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
	// BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
	// performance improvement of about 2x!
	hiprand_init(2020 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(Vector3 *fb, int max_x, int max_y, int ns, Camera **cam,	Hittable **world, hiprandState *rand_state)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;
	
	hiprandState local_rand_state = rand_state[pixel_index];
	Vector3 c(0.0f, 0.0f, 0.0f);
	for (int s = 0; s < ns; s++)
	{
		float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
		float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
		Ray r = (*cam)->getRay(u, v, &local_rand_state);
		c += ray_color(r, world, &local_rand_state);
	}
	
	rand_state[pixel_index] = local_rand_state;
	// gamma correction
	c = c / float(ns);
	c[0] = sqrt(c[0]);
	c[1] = sqrt(c[1]);
	c[2] = sqrt(c[2]);
	
	fb[pixel_index] = c;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(Hittable **d_list, Hittable **d_world, Camera** d_camera, int nx, int ny, hiprandState *rand_state)
{
	if(threadIdx.x == 0 && blockIdx.x == 0)
	{
		hiprandState local_rand_state = *rand_state;

		d_list[0] = new Sphere(Vector3(0, -1000.0f, -1), 1000.0f,
								new Lambertian(Vector3(0.5f, 0.5f, 0.5f)));
		int i = 1;
		for (int a = -11; a < 11; a++)
		{
			for (int b = -11; b < 11; b++)
			{
				float choose_mat = RND;
				Vector3 center(a + RND, 0.2f, b + RND);
				if (choose_mat < 0.8f)
				{
					d_list[i++] = new Sphere(center, 0.2f,
						new Lambertian(Vector3(RND * RND, RND * RND, RND * RND)));
				}
				else if (choose_mat < 0.95f)
				{
					d_list[i++] = new Sphere(center, 0.2f,
						new Metal(Vector3(0.5f * (1.0f + RND), 0.5f * (1.0f + RND), 0.5f * (1.0f + RND)), 0.5f * RND));
				}
				else
				{
					d_list[i++] = new Sphere(center, 0.2f, new Dielectric(1.5f));
				}
			}
		}
		d_list[i++] = new Sphere(Vector3(0, 1, 0), 1.0f, new Dielectric(1.5f));
		d_list[i++] = new Sphere(Vector3(-4, 1, 0), 1.0f, new Lambertian(Vector3(0.4f, 0.2f, 0.1f)));
		d_list[i++] = new Sphere(Vector3(4, 1, 0), 1.0f, new Metal(Vector3(0.7f, 0.6f, 0.5f), 0.0f));
		
		*rand_state = local_rand_state;
		*d_world = new HittableList(d_list, 22 * 22 + 1 + 3);

		Vector3 lookfrom(13, 2, 3);
		Vector3 lookat(0, 0, 0);
		float dist_to_focus = 10.0f; (lookfrom - lookat).getLength();
		float aperture = 0.1f;
		*d_camera = new Camera(
			lookfrom,
			lookat,
			Vector3(0, 1, 0),
			30.0f,
			float(nx) / float(ny),
			aperture,
			dist_to_focus
		);
	}
}

__global__ void free_world(Hittable **d_list, Hittable **d_world, Camera** d_camera)
{
	for (int i = 0; i < 22 * 22 + 1 + 3; i++)
	{
		delete ((Sphere*)d_list[i])->m_mat_ptr;
		delete d_list[i];
	}
	delete *d_world;
	delete *d_camera;
}

int main() {
	int nx = 1200;
	int ny = 600;
	int ns = 100;
	int tx = 8;
	int ty = 8;

	std::cerr << "Rendering a " << nx << "x" << ny << " image ";
	std::cerr << "in " << tx << "x" << ty << " blocks.\n";

	int num_pixels = nx * ny;
	size_t fb_size = num_pixels * sizeof(Vector3);

	// allocate FB
	Vector3 *fb;
	checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

	// allocate random state
	hiprandState *d_rand_state;
	checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));
	hiprandState *d_rand_state2;
	checkCudaErrors(hipMalloc((void**)&d_rand_state2, 1 * sizeof(hiprandState)));

	// we need that 2nd random state to be initialized for the world creation
	rand_init<<<1, 1>>>(d_rand_state2);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// make our world of hitables
	Hittable **d_list;
	int num_hitables = 22 * 22 + 1 + 3;
	checkCudaErrors(hipMalloc((void**)&d_list, num_hitables * sizeof(Hittable*)));
	Hittable **d_world; 
	checkCudaErrors(hipMalloc((void**)&d_world, sizeof(Hittable*)));
	Camera **d_camera;
	checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(Camera *)));
	create_world<<<1, 1>>>(d_list, d_world, d_camera, nx, ny, d_rand_state2);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	clock_t start, stop;
	start = clock();

	// Render our buffer
	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);

	render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	stop = clock();
	double timer_seconds_ = ((double)(stop - start)) / CLOCKS_PER_SEC;
	std::cerr << "took " << timer_seconds_ << " seconds to init render random.\n";
	start = clock();

	render<<<blocks, threads>>>(fb, nx, ny, ns, d_camera, d_world, d_rand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	stop = clock();
	double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
	std::cerr << "took " << timer_seconds << " seconds to render the image.\n";

	// Output FB as Image using stb_image
	unsigned char *image_buffer = new unsigned char[num_pixels * 4];
	for (int j = ny - 1; j >= 0; j--) {
		for (int i = 0; i < nx; i++) {
			size_t pixel_index = j * nx + i;
			image_buffer[4 * pixel_index + 0] = static_cast<unsigned char>(255.99f * clamp(fb[pixel_index].x, 0.0f, 0.999f));
			image_buffer[4 * pixel_index + 1] = static_cast<unsigned char>(255.99f * clamp(fb[pixel_index].y, 0.0f, 0.999f));
			image_buffer[4 * pixel_index + 2] = static_cast<unsigned char>(255.99f * clamp(fb[pixel_index].z, 0.0f, 0.999f));
			image_buffer[4 * pixel_index + 3] = static_cast<unsigned char>(255.99f * clamp(1                , 0.0f, 0.999f));
		}
	}

	stbi_flip_vertically_on_write(1);
	stbi_write_png(
		"./img_new.png",
		nx, 
		ny,
		4, 
		image_buffer,
		nx * 4
	);

	std::cerr << "\nDone.\n";

	// clean up
	delete[] image_buffer;
	checkCudaErrors(hipDeviceSynchronize());
	free_world<<<1, 1>>>(d_list, d_world, d_camera);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipFree(d_world));
	checkCudaErrors(hipFree(d_camera));
	checkCudaErrors(hipFree(d_list));
	checkCudaErrors(hipFree(d_rand_state));
	checkCudaErrors(hipFree(d_rand_state2));
	checkCudaErrors(hipFree(fb));
	
	// useful for cuda-memcheck --leak-check full
	hipDeviceReset();
}